
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 1000

__global__ void add(const int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    int ha[N];
    int hb[N];
    int *da;
    int *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    int a = 1;
    printf("%d", a);

    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    add<<<N, 1>>>(da, db);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }
    hipFree(da);
    hipFree(db);
    return 0;
}