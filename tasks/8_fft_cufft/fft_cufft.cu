#include <cstddef>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"

#include "fft_cufft.h"

#define CUDA_CALL(status)                                     \
  {                                                           \
    if (status != hipSuccess) {                              \
      std::cout << "error " << __FILE__ << ":" << __LINE__;   \
      std::cout << " " << hipGetErrorString(status) << "\n"; \
      std::exit(0);                                           \
    }                                                         \
  }

#define CUFFT_CALL(status)                                          \
  {                                                                 \
    if (status != HIPFFT_SUCCESS) {                                  \
      std::cout << "error " << __FILE__ << ":" << __LINE__ << "\n"; \
      std::exit(0);                                                 \
    }                                                               \
  }

std::vector<float> FftCUFFT(const std::vector<float>& input, int batch) {
  size_t size = input.size();
  size_t byte_count = size * sizeof(float);
  size_t batch_size = size / batch / 2;

  hipfftComplex* dev_input_mem = nullptr;
  std::vector<float> result(size);

  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_input_mem), byte_count));

  CUDA_CALL(hipMemcpy(dev_input_mem, input.data(), byte_count,
                       hipMemcpyHostToDevice));

  hipfftHandle plan;

  CUFFT_CALL(hipfftPlan1d(&plan, batch_size, HIPFFT_C2C, batch));
  CUFFT_CALL(hipfftExecC2C(plan, dev_input_mem, dev_input_mem, HIPFFT_FORWARD));
  CUFFT_CALL(hipfftExecC2C(plan, dev_input_mem, dev_input_mem, HIPFFT_BACKWARD));

  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(result.data()),
                       reinterpret_cast<void*>(dev_input_mem), byte_count,
                       hipMemcpyDeviceToHost));

  for (size_t i = 0; i < size; i++) {
    result[i] /= static_cast<float>(batch_size);
  }

  hipfftDestroy(plan);
  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_input_mem)));

  return result;
}
