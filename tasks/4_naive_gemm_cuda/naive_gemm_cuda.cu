#include "hip/hip_runtime.h"
#include <cmath>
#include <cstddef>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "naive_gemm_cuda.h"

#define CUDA_CALL(status)                                     \
  {                                                           \
    if (status != hipSuccess) {                              \
      std::cout << "error " << __FILE__ << ":" << __LINE__;   \
      std::cout << " " << hipGetErrorString(status) << "\n"; \
      std::exit(0);                                           \
    }                                                         \
  }

__global__ void naive_gemm_kernel(const float* a, const float* b, float* result,
                                  size_t n) {
  size_t j = blockIdx.x * blockDim.x + threadIdx.x;
  size_t i = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n) {
    float res = 0.0f;
    for (int k = 0; k < n; ++k) {
      res += a[i * n + k] * b[k * n + j];
    }
    result[i * n + j] = res;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, int n) {
  size_t size = n * n;
  size_t byte_count = size * sizeof(float);

  float* dev_a_mem = nullptr;
  float* dev_b_mem = nullptr;

  std::vector<float> result(size);
  float* dev_result_mem = nullptr;

  constexpr size_t block_dim_value = 32;  // max 1024 threads per block
  dim3 block_dim(block_dim_value, block_dim_value);
  dim3 grid_dim(ceil(n / block_dim.x), ceil(n / block_dim.y));

  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_a_mem), byte_count));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_b_mem), byte_count));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_result_mem), byte_count));

  CUDA_CALL(
      hipMemcpy(dev_a_mem, a.data(), byte_count, hipMemcpyHostToDevice));
  CUDA_CALL(
      hipMemcpy(dev_b_mem, b.data(), byte_count, hipMemcpyHostToDevice));

  naive_gemm_kernel<<<grid_dim, block_dim>>>(dev_a_mem, dev_b_mem,
                                             dev_result_mem, n);
  CUDA_CALL(hipGetLastError());

  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(result.data()),
                       reinterpret_cast<void*>(dev_result_mem), byte_count,
                       hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_a_mem)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_b_mem)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_result_mem)));

  return result;
}
