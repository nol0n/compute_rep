#include <cstddef>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#include "gemm_cublas.h"

#define CUDA_CALL(status)                                     \
  {                                                           \
    if (status != hipSuccess) {                              \
      std::cout << "error " << __FILE__ << ":" << __LINE__;   \
      std::cout << " " << hipGetErrorString(status) << "\n"; \
      std::exit(0);                                           \
    }                                                         \
  }

#define CUBLAS_CALL(status)                                         \
  {                                                                 \
    if (status != HIPBLAS_STATUS_SUCCESS) {                          \
      std::cout << "error " << __FILE__ << ":" << __LINE__ << "\n"; \
      std::exit(0);                                                 \
    }                                                               \
  }

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, int n) {
  size_t size = n * n;
  size_t byte_count = size * sizeof(float);

  float* dev_a_mem = nullptr;
  float* dev_b_mem = nullptr;

  std::vector<float> result(size);
  float* dev_result_mem = nullptr;
  hipblasHandle_t handle;

  float alpha = 1.0f;
  float beta = 0.0f;

  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_a_mem), byte_count));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_b_mem), byte_count));
  CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&dev_result_mem), byte_count));

  CUDA_CALL(
      hipMemcpy(dev_a_mem, a.data(), byte_count, hipMemcpyHostToDevice));
  CUDA_CALL(
      hipMemcpy(dev_b_mem, b.data(), byte_count, hipMemcpyHostToDevice));

  CUBLAS_CALL(hipblasCreate(&handle));
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, dev_a_mem, n, dev_b_mem, n, &beta, dev_result_mem, n));
  hipblasDestroy(handle);

  CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(result.data()),
                       reinterpret_cast<void*>(dev_result_mem), byte_count,
                       hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_a_mem)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_b_mem)));
  CUDA_CALL(hipFree(reinterpret_cast<void*>(dev_result_mem)));

  return result;
}
